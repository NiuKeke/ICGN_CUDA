#include "hip/hip_runtime.h"
#include "disp_optimize_icgn.cuh"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include ""
#define BLOCK_DATA_DIM_X 32
#define BLOCK_DATA_DIM_Y 32
#define BLOCK_THREAD_DIM_X 8
#define BLOCK_THREAD_DIM_Y 8
#define NUM_PER_THREAD_X 4
#define NUM_PER_THREAD_Y 4
__global__ void generate_gradient_image_kernel(int width, int height, uchar *_src_image,
                                               float *_x_gradient_image, float *_y_gradient_image)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * width + x;
    if (x >= width || y >= height)
    {
        return;
    }

    float result = 0.0f;
    if ((x + 2) >= width || (x - 2) < 0)
    {
        result = 0.0f;
    }
    else
    {
        result -= (float)_src_image[y * width + x + 2] * 0.083333333333333f;
        result += (float)_src_image[y * width + x + 1] * 0.666666666666667f;
        result -= (float)_src_image[y * width + x - 1] * 0.666666666666667f;
        result += (float)_src_image[y * width + x - 2] * 0.083333333333333f;
    }
    _y_gradient_image[index] = result;
    result = 0.0f;
    if ((y + 2) >= height || (y - 2) < 0)
    {
        result = 0.0f;
    }
    else
    {
        result -= (float)_src_image[(y + 2) * width + x] * 0.083333333333333f;
        result += (float)_src_image[(y + 1) * width + x] * 0.666666666666667f;
        result -= (float)_src_image[(y - 1) * width + x] * 0.666666666666667f;
        result += (float)_src_image[(y - 2) * width + x] * 0.083333333333333f;
    }
    _x_gradient_image[index] = result;
}

__global__ void calHessianMat_kernel(int subset, int sideW, int width, int height, float *_x_grad_image, float *_y_grad_image,
                                     double *_hessian_mat)
{
    int g_x = blockIdx.x * blockDim.x * NUM_PER_THREAD_X + threadIdx.x;
    int g_y = blockIdx.y * blockDim.y * NUM_PER_THREAD_Y + threadIdx.y;
    int thread_index = threadIdx.y * blockDim.x + threadIdx.x;

    int halfSubset = subset / 2;
    int halfWinSize = halfSubset + sideW; // 7+5;
    g_x = (g_x - 2 * blockIdx.x * halfWinSize) < 0 ? 0 : (g_x - 2 * blockIdx.x * halfWinSize);
    g_y = (g_y - 2 * blockIdx.y * halfWinSize) < 0 ? 0 : (g_y - 2 * blockIdx.y * halfWinSize);

    // if (blockIdx.y == 0 && blockIdx.x == 38)
    // {
    //     //printf("blockIdx.x: %d, blockIdx.y: %d, g_x: %d, g_y: %d\n", blockIdx.x, blockIdx.y, g_x, g_y);
    // }
    __shared__ float _x_grad_image_sm[BLOCK_DATA_DIM_X * BLOCK_DATA_DIM_Y]; // 4k
    __shared__ float _y_grad_image_sm[BLOCK_DATA_DIM_X * BLOCK_DATA_DIM_Y]; // 4k
    for (int i = 0; i < NUM_PER_THREAD_Y; i++)
    {
        for (int j = 0; j < NUM_PER_THREAD_X; j++)
        {
            _x_grad_image_sm[(threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X] =
                _x_grad_image[(g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X];

            _y_grad_image_sm[(threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X] =
                _y_grad_image[(g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X];

            // if (blockIdx.y == 0 && blockIdx.x == 38)
            // {
            //     int index_region = (threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X;
            //     int row_region = index_region / 32;
            //     int col_region = index_region % 32;
            //     // if (row_region == 18)
            //     // {
            //     //     printf("threadIdx.x: %d, threadIdx.y: %d, row: %d, col: %d, blockIdx.x: %d, blockIdx.y: %d, g_x: %d, g_y: %d,row_region: %d, col_region: %d, _x_grad_image_sm[%d]:%lf, _x_grad_image[%d]: %lf\n",
            //     //            threadIdx.x, threadIdx.y, i, j, blockIdx.x, blockIdx.y, g_x, g_y, row_region, col_region,
            //     //            (threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X,
            //     //            _x_grad_image_sm[(threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X],
            //     //            (g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X,
            //     //            _x_grad_image[(g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X]);
            //     // }
            // }
        }
    }
    __syncthreads();
    // for (int j = -halfSubset; j <= halfSubset; j++) // y
    // {
    //     for (int k = -halfSubset; k <= halfSubset; k++) // x
    //     {
    //         if (blockIdx.y == 0 && blockIdx.x == 40)
    //         {
    //             printf("threadIdx.x: %d, threadIdx.y: %d, row: %d, col: %d, blockIdx.x: %d, blockIdx.y: %d, g_x: %d, g_y: %d,_x_grad_image_sm[%d]:%lf, _y_grad_image_sm[%d]: %lf\n",
    //                    threadIdx.x, threadIdx.y, j, k, blockIdx.x, blockIdx.y, g_x, g_y,
    //                    halfWinSize * BLOCK_THREAD_DIM_X * NUM_PER_THREAD_X + threadIdx.y * BLOCK_THREAD_DIM_X + j + halfWinSize + k + threadIdx.x,
    //                    _x_grad_image_sm[halfWinSize * BLOCK_THREAD_DIM_X * NUM_PER_THREAD_X + threadIdx.y * BLOCK_THREAD_DIM_X + j + halfWinSize + k + threadIdx.x],
    //                    halfWinSize * BLOCK_THREAD_DIM_X * NUM_PER_THREAD_X + threadIdx.y * BLOCK_THREAD_DIM_X + j + halfWinSize + k + threadIdx.x,
    //                    _y_grad_image_sm[halfWinSize * BLOCK_THREAD_DIM_X * NUM_PER_THREAD_X + threadIdx.y * BLOCK_THREAD_DIM_X + j + halfWinSize + k + threadIdx.x]);
    //         }
    //     }
    // }

    double hessian[6 * 6] = {0};
    // if ((g_x - halfWinSize) >= 0 && (g_x + halfWinSize) < width && (g_y - halfWinSize) >= 0 && (g_y + halfWinSize) < height)
    {
        for (int j = -halfSubset; j <= halfSubset; j++) // y
        {
            for (int k = -halfSubset; k <= halfSubset; k++) // x
            {
                double Jacobian[6];
                Jacobian[0] = _x_grad_image_sm[(halfWinSize + threadIdx.y + k) * BLOCK_THREAD_DIM_Y * NUM_PER_THREAD_Y + halfWinSize + j + threadIdx.x];
                Jacobian[1] = Jacobian[0] * double(k) / double(halfSubset + 1);//x;
                Jacobian[2] = Jacobian[0] * double(j) / double(halfSubset + 1);//y;
                Jacobian[3] = _y_grad_image_sm[(halfWinSize + threadIdx.y + k) * BLOCK_THREAD_DIM_X * NUM_PER_THREAD_X + halfWinSize + j + threadIdx.x];
                Jacobian[4] = Jacobian[3] * double(k) / double(halfSubset + 1);
                Jacobian[5] = Jacobian[3] * double(j) / double(halfSubset + 1);

                // if (blockIdx.y == 0 && blockIdx.x == 38)
                // {
                //     int index_region = (halfWinSize + threadIdx.y + k) * BLOCK_THREAD_DIM_Y * NUM_PER_THREAD_Y + halfWinSize + j + threadIdx.x;
                //     int row_region = index_region / 32;
                //     int col_region = index_region % 32;
                //     if (row_region == 18 && threadIdx.x == 5 && threadIdx.y == 1)
                //     {
                //         // printf("111 threadIdx.x: %d, threadIdx.y: %d, row: %d, col: %d, blockIdx.x: %d, blockIdx.y: %d, g_x: %d, g_y: %d,row_region: %d, col_region: %d, _x_grad_image_sm[%d]:%lf, _y_grad_image_sm[%d]: %lf\n",
                //         //        threadIdx.x, threadIdx.y, k, j, blockIdx.x, blockIdx.y, g_x, g_y, row_region, col_region,
                //         //        index_region,
                //         //        _x_grad_image_sm[index_region],
                //         //        index_region,
                //         //        _y_grad_image_sm[index_region]);
                //         // printf("Jacobian[0]: %lf, Jacobian[1]: %lf,Jacobian[2]: %lf,Jacobian[3]: %lf,Jacobian[4]: %lf,Jacobian[5]: %lf, localSub_x:%lf,localSub_y: %lf\n",
                //         //  Jacobian[0], Jacobian[1], Jacobian[2],
                //         //  Jacobian[3], Jacobian[4], Jacobian[5], float(k) / float(halfSubset + 1), float(j) / float(halfSubset + 1));
                //     }
                // }

                hessian[0] += Jacobian[0] * Jacobian[0];
                hessian[1] += Jacobian[0] * Jacobian[1];
                hessian[2] += Jacobian[0] * Jacobian[2];
                hessian[3] += Jacobian[0] * Jacobian[3];
                hessian[4] += Jacobian[0] * Jacobian[4];
                hessian[5] += Jacobian[0] * Jacobian[5];
                hessian[6] += Jacobian[1] * Jacobian[0];
                hessian[7] += Jacobian[1] * Jacobian[1];
                hessian[8] += Jacobian[1] * Jacobian[2];
                hessian[9] += Jacobian[1] * Jacobian[3];
                hessian[10] += Jacobian[1] * Jacobian[4];
                hessian[11] += Jacobian[1] * Jacobian[5];
                hessian[12] += Jacobian[2] * Jacobian[0];
                hessian[13] += Jacobian[2] * Jacobian[1];
                hessian[14] += Jacobian[2] * Jacobian[2];
                hessian[15] += Jacobian[2] * Jacobian[3];
                hessian[16] += Jacobian[2] * Jacobian[4];
                hessian[17] += Jacobian[2] * Jacobian[5];
                hessian[18] += Jacobian[3] * Jacobian[0];
                hessian[19] += Jacobian[3] * Jacobian[1];
                hessian[20] += Jacobian[3] * Jacobian[2];
                hessian[21] += Jacobian[3] * Jacobian[3];
                hessian[22] += Jacobian[3] * Jacobian[4];
                hessian[23] += Jacobian[3] * Jacobian[5];
                hessian[24] += Jacobian[4] * Jacobian[0];
                hessian[25] += Jacobian[4] * Jacobian[1];
                hessian[26] += Jacobian[4] * Jacobian[2];
                hessian[27] += Jacobian[4] * Jacobian[3];
                hessian[28] += Jacobian[4] * Jacobian[4];
                hessian[29] += Jacobian[4] * Jacobian[5];
                hessian[30] += Jacobian[5] * Jacobian[0];
                hessian[31] += Jacobian[5] * Jacobian[1];
                hessian[32] += Jacobian[5] * Jacobian[2];
                hessian[33] += Jacobian[5] * Jacobian[3];
                hessian[34] += Jacobian[5] * Jacobian[4];
                hessian[35] += Jacobian[5] * Jacobian[5];
            }
        }
    }

    __syncthreads();

    for (int i = 0; i < 6 * 6; i++)
    {
        _hessian_mat[(g_y + halfWinSize) * width + g_x + halfWinSize + i * width * height] = hessian[i];

        // if (blockIdx.y == 0 && blockIdx.x == 38 && threadIdx.x == 5 && threadIdx.y == 1)
        // {
        //     int hessian_index = (g_y + halfWinSize) * width + g_x + halfWinSize + i * width * height;
        //     printf("blockIdx.x: %d, blockIdx.y: %d, g_x: %d, g_y: %d, hessian_index: %d,hessian[%d]: %lf\n", blockIdx.x, blockIdx.y, g_x, g_y,
        //            hessian_index,i, hessian[i]);
        // }
    }
}
void CDispOptimizeICGN_GPU::run(cv::Mat &_l_image, cv::Mat &_r_image, cv::Mat &_src_disp, int subset, int sideW, int maxIter, cv::Mat &_result)
{
    // 生成左图像梯度影像,分为x,y两个方向;
    cv::Mat _x_gradient_image_cpu, _y_gradient_image_cpu;
    _x_gradient_image_cpu.create(_l_image.size(), CV_32FC1);
    _y_gradient_image_cpu.create(_l_image.size(), CV_32FC1);
    // generate_gradient_image(_l_image, _x_gradient_image, _y_gradient_image);
    // // 保存梯度影像;

    float *_x_gradient_image = nullptr;
    float *_y_gradient_image = nullptr;
    generate_gradient_image(_l_image, _x_gradient_image, _y_gradient_image);

    hipMemcpy(_x_gradient_image_cpu.data, _x_gradient_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(_y_gradient_image_cpu.data, _y_gradient_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    cv::imwrite("x_gradient_image_cpu.tif", _x_gradient_image_cpu);
    cv::imwrite("y_gradient_image_cpu.tif", _y_gradient_image_cpu);

    double *hessian = nullptr;
    generate_hessian_mat(subset, sideW, maxIter, _l_image.cols, _l_image.rows, _x_gradient_image, _y_gradient_image, hessian);

    cv::Mat hessianMat = cv::Mat(36, _l_image.rows * _l_image.cols, CV_64FC1);
    hipMemcpy(hessianMat.data, hessian, _l_image.rows * _l_image.cols * sizeof(double) * 36,
               hipMemcpyDeviceToHost);
    for (int n = 0; n < 36; n++)
    {

        double value = hessianMat.data[n * _l_image.rows * _l_image.cols + 16961];
        //printf("n: %d, value: %lf\n", n, value);
    }
    cv::imwrite("./hessian.tif", hessianMat);
}

void CDispOptimizeICGN_GPU::generate_hessian_mat(int subset, int sideW, int maxIter, int width, int height, float *_x_gradient_image,
                                                 float *_y_gradient_image, double *&_hessian_mat)
{
    hipMalloc((void **)&_hessian_mat, width * height * sizeof(double) * 6 * 6);

    int halfSubset = subset / 2;
    int halfWinSize = halfSubset + sideW; // 7+5;

    dim3 threads(8, 8);
    dim3 blocks((width - 2 * halfWinSize + threads.x - 1) / (threads.x),
                (height - 2 * halfWinSize + threads.y - 1) / (threads.y));

    printf("width: %d, height: %d, blocks.x: %d, blocks.y: %d, threads.x: %d, threads.y: %d\n",
           width, height, blocks.x, blocks.y, threads.x, threads.y);
    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    calHessianMat_kernel<<<blocks, threads>>>(subset, sideW, width, height, _x_gradient_image, _y_gradient_image, _hessian_mat);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_hessian_mat time: %f ms\n", time);
}

void CDispOptimizeICGN_GPU::generate_gradient_image(cv::Mat &_l_image, float *&_x_gradient_image, float *&_y_gradient_image)
{
    uchar *src_image = nullptr;
    hipMalloc((void **)&src_image, _l_image.rows * _l_image.cols * sizeof(uchar));
    hipMemcpy(src_image, _l_image.data, _l_image.rows * _l_image.cols * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((_l_image.cols + threads.x - 1) / threads.x, (_l_image.rows + threads.y - 1) / threads.y);

    hipMalloc((void **)&_x_gradient_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipMalloc((void **)&_y_gradient_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    generate_gradient_image_kernel<<<blocks, threads>>>(_l_image.cols, _l_image.rows, src_image, _x_gradient_image, _y_gradient_image);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_gradient_image_x time: %f ms\n", time);

    return;
}

void CDispOptimizeICGN_GPU::generate_gradient_image(cv::Mat &_l_image, cv::Mat &_x_gradient_image, cv::Mat &_y_gradient_image)
{
    uchar *src_image = nullptr;
    hipMalloc((void **)&src_image, _l_image.rows * _l_image.cols * sizeof(uchar));
    hipMemcpy(src_image, _l_image.data, _l_image.rows * _l_image.cols * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((_l_image.cols + threads.x - 1) / threads.x, (_l_image.rows + threads.y - 1) / threads.y);
    float *_x_dst_image = nullptr;
    hipMalloc((void **)&_x_dst_image, _l_image.rows * _l_image.cols * sizeof(float));
    float *_y_dst_image = nullptr;
    hipMalloc((void **)&_y_dst_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    generate_gradient_image_kernel<<<blocks, threads>>>(_l_image.cols, _l_image.rows, src_image, _x_dst_image, _y_dst_image);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_gradient_image_x time: %f ms\n", time);
    hipMemcpy(_x_gradient_image.data, _x_dst_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(_y_gradient_image.data, _y_dst_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);

    hipFree(src_image);
    hipFree(_x_dst_image);
    hipFree(_y_dst_image);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return;
}