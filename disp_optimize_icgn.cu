#include "hip/hip_runtime.h"
#include "disp_optimize_icgn.cuh"
#include <opencv2/opencv.hpp>
__global__ void generate_gradient_image_kernel(int width, int height, uchar *_src_image,
                                               float *_x_gradient_image, float *_y_gradient_image)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * width + x;
    if (x >= width || y >= height)
    {
        return;
    }

    float result = 0.0f;
    if ((x + 2) >= width || (x - 2) < 0)
    {
        result = 0.0f;
    }
    else
    {
        result -= (float)_src_image[y * width + x + 2] * 0.083333333333333f;
        result += (float)_src_image[y * width + x + 1] * 0.666666666666667f;
        result -= (float)_src_image[y * width + x - 1] * 0.666666666666667f;
        result += (float)_src_image[y * width + x - 2] * 0.083333333333333f;
    }
    _x_gradient_image[index] = result;
    if ((y + 2) >= height || (y - 2) < 0)
    {
        result = 0.0f;
    }
    else
    {
        result -= (float)_src_image[(y + 2) * width + x] * 0.083333333333333f;
        result += (float)_src_image[(y + 1) * width + x] * 0.666666666666667f;
        result -= (float)_src_image[(y - 1) * width + x] * 0.666666666666667f;
        result += (float)_src_image[(y - 2) * width + x] * 0.083333333333333f;
    }
    _y_gradient_image[index] = result;
}

__global__ void calHessianMat(int subset, int sideW, int width, int height, float *_x_grad_image, float *_y_grad_image, float *_hessian_mat)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int halfSubset = subset / 2;
    int halfWinSize = halfSubset + sideW;//7+5;

    if((x - halfWinSize) < 0 || (x + halfWinSize) > width || (y - halfWinSize) < 0 || (y + halfWinSize) > height){
        return ;
    }

    
}
void CDispOptimizeICGN_GPU::run(cv::Mat &_l_image, cv::Mat &_r_image, cv::Mat &_src_disp, int subset, int sideW, int maxIter, cv::Mat &_result)
{
    // 生成左图像梯度影像,分为x,y两个方向;
    cv::Mat _x_gradient_image, _y_gradient_image;
    _x_gradient_image.create(_l_image.size(), CV_32FC1);
    _y_gradient_image.create(_l_image.size(), CV_32FC1);
    generate_gradient_image(_l_image, _x_gradient_image, _y_gradient_image);
    // 保存梯度影像;
    cv::imwrite("x_gradient_image_gpu.jpg", _x_gradient_image);
    cv::imwrite("y_gradient_image_gpu.jpg", _y_gradient_image);
}

void CDispOptimizeICGN_GPU::generate_gradient_image(cv::Mat &_l_image, cv::Mat &_x_gradient_image, cv::Mat &_y_gradient_image)
{
    uchar *src_image = nullptr;
    hipMalloc((void **)&src_image, _l_image.rows * _l_image.cols * sizeof(uchar));
    hipMemcpy(src_image, _l_image.data, _l_image.rows * _l_image.cols * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((_l_image.cols + threads.x - 1) / threads.x, (_l_image.rows + threads.y - 1) / threads.y);
    float *_x_dst_image = nullptr;
    hipMalloc((void **)&_x_dst_image, _l_image.rows * _l_image.cols * sizeof(float));
    float *_y_dst_image = nullptr;
    hipMalloc((void **)&_y_dst_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    generate_gradient_image_kernel<<<blocks, threads>>>(_l_image.cols, _l_image.rows, src_image, _x_dst_image, _y_dst_image);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_gradient_image_x time: %f ms\n", time);
    hipMemcpy(_x_gradient_image.data, _x_dst_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(_y_gradient_image.data, _y_dst_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);

    hipFree(src_image);
    hipFree(_x_dst_image);
    hipFree(_y_dst_image);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return;
}