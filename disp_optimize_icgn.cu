#include "hip/hip_runtime.h"
#include "disp_optimize_icgn.cuh"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "cuda_device_runtime_api.h"
#include ""
#define BLOCK_DATA_DIM_X 32
#define BLOCK_DATA_DIM_Y 32
#define BLOCK_THREAD_DIM_X 8
#define BLOCK_THREAD_DIM_Y 8
#define NUM_PER_THREAD_X 4
#define NUM_PER_THREAD_Y 4
__global__ void generate_gradient_image_kernel(int width, int height, uchar *_src_image,
                                               float *_x_gradient_image, float *_y_gradient_image)
{

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * width + x;
    if (x >= width || y >= height)
    {
        return;
    }

    float result = 0.0f;
    if ((x + 2) >= width || (x - 2) < 0)
    {
        result = 0.0f;
    }
    else
    {
        result -= (float)_src_image[y * width + x + 2] * 0.083333333333333f;
        result += (float)_src_image[y * width + x + 1] * 0.666666666666667f;
        result -= (float)_src_image[y * width + x - 1] * 0.666666666666667f;
        result += (float)_src_image[y * width + x - 2] * 0.083333333333333f;
    }
    _x_gradient_image[index] = result;
    if ((y + 2) >= height || (y - 2) < 0)
    {
        result = 0.0f;
    }
    else
    {
        result -= (float)_src_image[(y + 2) * width + x] * 0.083333333333333f;
        result += (float)_src_image[(y + 1) * width + x] * 0.666666666666667f;
        result -= (float)_src_image[(y - 1) * width + x] * 0.666666666666667f;
        result += (float)_src_image[(y - 2) * width + x] * 0.083333333333333f;
    }
    _y_gradient_image[index] = result;
}

__global__ void calHessianMat_kernel(int subset, int sideW, int width, int height, float *_x_grad_image, float *_y_grad_image,
                                     float *_hessian_mat)
{
    int g_x = blockIdx.x * blockDim.x * NUM_PER_THREAD_X + threadIdx.x;
    int g_y = blockIdx.y * blockDim.y * NUM_PER_THREAD_Y + threadIdx.y;
    int thread_index = threadIdx.y * blockDim.x + threadIdx.x;

    int halfSubset = subset / 2;
    int halfWinSize = halfSubset + sideW; // 7+5;
    __shared__ float _x_grad_image_sm[BLOCK_DATA_DIM_X * BLOCK_DATA_DIM_Y];
    __shared__ float _y_grad_image_sm[BLOCK_DATA_DIM_X * BLOCK_DATA_DIM_Y];
    for (int i = 0; i < NUM_PER_THREAD_Y; i++)
    {
        for (int j = 0; j < NUM_PER_THREAD_X; j++)
        {
            _x_grad_image_sm[(threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X] =
                _x_grad_image[(g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X];

            _y_grad_image_sm[(threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X] =
                _y_grad_image[(g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X];
        }
    }

    // for (int i = 0; i < NUM_PER_THREAD_Y; i++)
    // {
    //     for (int j = 0; j < NUM_PER_THREAD_X; j++)
    //     {
    //         if (blockIdx.x == 0 && blockIdx.y == 0 && i == 0 &&j == 0){
    //             printf("threadX:%d, threadY:%d, i: %d,j: %d,_x_grad_image_sm[%d]:%lf, g_x: %d. g_y: %d,g_image[%d]: %lf\n",
    //                threadIdx.x, threadIdx.y, i,j,
    //                (threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X,
    //                _x_grad_image_sm[(threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X],
    //                g_x,g_y,
    //                (g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X,
    //                _x_grad_image[(g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X]);
    //         }      
    //         // if (blockIdx.x == 0 && blockIdx.y == 1 && threadIdx.x == 0 && threadIdx.y == 0)
    //         // {
    //         //     printf("threadX:%d, threadY:%d, i:%d, j:%d, sm_index:%d, g_x:%d, g_y:%d, g_index1:%d \n",
    //         //            threadIdx.x, threadIdx.y, i, j,
    //         //            (threadIdx.y + i * BLOCK_THREAD_DIM_Y) * blockDim.x * NUM_PER_THREAD_X + threadIdx.x + j * BLOCK_THREAD_DIM_X,
    //         //            g_x, g_y,
    //         //            (g_y + i * BLOCK_THREAD_DIM_Y) * width + g_x + j * BLOCK_THREAD_DIM_X);
    //         // }
    //     }
    // }

    __syncthreads();

    if ((g_x - halfWinSize) >= 0 && (g_x + halfWinSize) < width && (g_y - halfWinSize) >= 0 && (g_y + halfWinSize) < height)
    {
        
    }
}
void CDispOptimizeICGN_GPU::run(cv::Mat &_l_image, cv::Mat &_r_image, cv::Mat &_src_disp, int subset, int sideW, int maxIter, cv::Mat &_result)
{
    // 生成左图像梯度影像,分为x,y两个方向;
    cv::Mat _x_gradient_image_cpu, _y_gradient_image_cpu;
    _x_gradient_image_cpu.create(_l_image.size(), CV_32FC1);
    _y_gradient_image_cpu.create(_l_image.size(), CV_32FC1);
    // generate_gradient_image(_l_image, _x_gradient_image, _y_gradient_image);
    // // 保存梯度影像;
    

    float *_x_gradient_image = nullptr;
    float *_y_gradient_image = nullptr;
    generate_gradient_image(_l_image, _x_gradient_image, _y_gradient_image);

    hipMemcpy(_x_gradient_image_cpu.data, _x_gradient_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(_y_gradient_image_cpu.data, _y_gradient_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    cv::imwrite("x_gradient_image_cpu.tif", _x_gradient_image_cpu);
    cv::imwrite("y_gradient_image_cpu.tif", _y_gradient_image_cpu);

    float *hessian = nullptr;
    generate_hessian_mat(subset, sideW, maxIter, _l_image.cols, _l_image.rows, _x_gradient_image, _y_gradient_image, hessian);

    cv::Mat hessianMat = cv::Mat(_l_image.rows, _l_image.cols * 36, CV_32FC1);
    hipMemcpy(hessianMat.data, hessian, _l_image.rows * _l_image.cols * sizeof(float) * 36,
               hipMemcpyDeviceToHost);
    cv::imwrite("./hessian.tif", hessianMat);
}

void CDispOptimizeICGN_GPU::generate_hessian_mat(int subset, int sideW, int maxIter, int width, int height, float *_x_gradient_image,
                                                 float *_y_gradient_image, float *_hessian_mat)
{
    hipMalloc((void **)&_hessian_mat, width * height * sizeof(float) * 6 * 6);

    dim3 threads(8, 8);
    dim3 blocks((width + threads.x * NUM_PER_THREAD_X - 1) / (threads.x * NUM_PER_THREAD_X),
                (height + threads.y * NUM_PER_THREAD_Y - 1) / (threads.y * NUM_PER_THREAD_Y));

    printf("width: %d, height: %d, blocks.x: %d, blocks.y: %d, threads.x: %d, threads.y: %d\n",
           width, height, blocks.x, blocks.y, threads.x, threads.y);
    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    calHessianMat_kernel<<<blocks, threads>>>(subset, sideW, width, height, _x_gradient_image, _y_gradient_image, _hessian_mat);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_hessian_mat time: %f ms\n", time);
}

void CDispOptimizeICGN_GPU::generate_gradient_image(cv::Mat &_l_image, float *&_x_gradient_image, float *&_y_gradient_image)
{
    uchar *src_image = nullptr;
    hipMalloc((void **)&src_image, _l_image.rows * _l_image.cols * sizeof(uchar));
    hipMemcpy(src_image, _l_image.data, _l_image.rows * _l_image.cols * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((_l_image.cols + threads.x - 1) / threads.x, (_l_image.rows + threads.y - 1) / threads.y);

    hipMalloc((void **)&_x_gradient_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipMalloc((void **)&_y_gradient_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    generate_gradient_image_kernel<<<blocks, threads>>>(_l_image.cols, _l_image.rows, src_image, _x_gradient_image, _y_gradient_image);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_gradient_image_x time: %f ms\n", time);

    return;
}

void CDispOptimizeICGN_GPU::generate_gradient_image(cv::Mat &_l_image, cv::Mat &_x_gradient_image, cv::Mat &_y_gradient_image)
{
    uchar *src_image = nullptr;
    hipMalloc((void **)&src_image, _l_image.rows * _l_image.cols * sizeof(uchar));
    hipMemcpy(src_image, _l_image.data, _l_image.rows * _l_image.cols * sizeof(uchar),
               hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((_l_image.cols + threads.x - 1) / threads.x, (_l_image.rows + threads.y - 1) / threads.y);
    float *_x_dst_image = nullptr;
    hipMalloc((void **)&_x_dst_image, _l_image.rows * _l_image.cols * sizeof(float));
    float *_y_dst_image = nullptr;
    hipMalloc((void **)&_y_dst_image, _l_image.rows * _l_image.cols * sizeof(float));

    hipEvent_t start, stop;
    float time = 0.0f;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    generate_gradient_image_kernel<<<blocks, threads>>>(_l_image.cols, _l_image.rows, src_image, _x_dst_image, _y_dst_image);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipDeviceSynchronize();
    printf("generate_gradient_image_x time: %f ms\n", time);
    hipMemcpy(_x_gradient_image.data, _x_dst_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(_y_gradient_image.data, _y_dst_image, _l_image.rows * _l_image.cols * sizeof(float),
               hipMemcpyDeviceToHost);

    hipFree(src_image);
    hipFree(_x_dst_image);
    hipFree(_y_dst_image);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return;
}